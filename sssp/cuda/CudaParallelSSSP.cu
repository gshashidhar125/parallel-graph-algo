#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaGraph.h"
#define cprint(...) printMutex.lock(); printf(__VA_ARGS__); printMutex.unlock();
#define p(x) printMutex.lock(); cout << x << endl; printMutex.unlock();

#define CUDA_ERR_CHECK  \
if( err != hipSuccess) { \
    printf("CUDA error: %s ** at Line %d\n", hipGetErrorString(err), __LINE__); \
    return EXIT_FAILURE; \
} \

using namespace std;

__device__ int *graph[2], *worklist, d_tail;
__device__ bool d_terminate;
__global__ void CudaInitialize(int *vertexArray, int *edgeArray) {

    graph[0] = vertexArray;
    graph[1] = edgeArray;
}
__global__ void Cuda_SSSP(CudaGraphClass *graphData) {
    
    printf("BlockId = %d, Thread ID : %d\n", blockIdx.x, threadIdx.x);
    int tId = blockIdx.x * blockDim.x + threadIdx.x;
}

void CudaGraphClass::callSSSP() {

    print("Hello inside cuda code\n");
    copyGraphToDevice();
    
    int terminate = false;
    while (terminate == false) {
        terminate = true;
//        outs("Queue: Head: %d, Tail: %d\n", currentQueueHead, currentQueueTail);
        hipMemcpyToSymbol(HIP_SYMBOL(d_terminate), &terminate, sizeof(bool), 0, hipMemcpyHostToDevice);
//        hipMemcpyToSymbol(HIP_SYMBOL(d_tail), &tail, sizeof(bool), 0, hipMemcpyHostToDevice);
        Cuda_SSSP<<<2, 5>>>(this);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(d_terminate), sizeof(bool), 0, hipMemcpyDeviceToHost);
    }
}
int CudaGraphClass::copyGraphToDevice() {

    int *vertexArray, *edgeArray;
    hipError_t err;
    err = hipMalloc((void **)&vertexArray, (numVertices + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&edgeArray, (numEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemcpy(vertexArray, row[0], (numVertices + 2) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    err = hipMemcpy(edgeArray, row[1], (numEdges + 1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    CudaInitialize<<<1, 1>>>(edgeArray, vertexArray);
    hipDeviceSynchronize();
    return 0;
}
