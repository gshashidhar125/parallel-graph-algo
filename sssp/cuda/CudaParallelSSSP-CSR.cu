#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaGraph.h"
#define cprint(...) printMutex.lock(); printf(__VA_ARGS__); printMutex.unlock();
#define p(x) printMutex.lock(); cout << x << endl; printMutex.unlock();

#define CUDA_ERR_CHECK  \
if( err != hipSuccess) { \
    printf("CUDA error: %s ** at Line %d\n", hipGetErrorString(err), __LINE__); \
    return EXIT_FAILURE; \
}

#define CUDA_SET_DEVICE_ID \
hipSetDevice(0);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

using namespace std;

__device__ int *graph[3], d_numVertices, d_numEdges, *d_worklist, *d_gatherWorklist, d_worklistLength;
__device__ int *d_prefixSum, *d_blockPrefixSum;
__device__ int *d_prefixLevel;
__device__ bool d_terminate;

__global__ void CudaInitialize(int *vertexArray, int *edgeArray, int *weightArray, int *worklist, int *worklist2, int *prefixSum, int *blockPrefixSum, int *prefixLevel, int numVertices, int numEdges) {

    d_numVertices = numVertices;
    d_numEdges = numEdges;
    graph[0] = vertexArray;
    graph[1] = edgeArray;
    graph[2] = weightArray;
    d_worklist = worklist;
    d_gatherWorklist = worklist2;
    d_prefixSum = prefixSum;
    d_blockPrefixSum = blockPrefixSum;
    d_prefixLevel = prefixLevel;
    for (int i = 0; i < numVertices; i++) {
        d_worklist[i] = i;
    }
    d_worklistLength = d_numVertices;
    print("WLLenght = %d, numVertices = %d\n", d_worklistLength, d_numVertices);
    d_prefixSum[numVertices + 1] = 0;
    d_prefixSum[numVertices + 2] = 0;
}
__global__ void CudaPrintGraph() {

    print("Vertex Array:\n");
    for (int i = 0; i < d_numVertices + 2; i++)
        print(" %d", graph[0][i]);
    print("\n");
    print("Edge Array:\n");
    for (int i = 0; i < d_numEdges + 1; i++)
        print(" %d[%d]", graph[1][i], graph[2][i]);
    print("\n");
}


// Prefix Sum calculation within a single block
__global__ void Cuda_IntraBlockPrefixSum() {
    
    extern __shared__ int temp[];
    __shared__ int blockPrefixSum;
    int vertex, maxLength, numNeighbours, tId = threadIdx.x;
    if ((d_worklistLength - blockIdx.x * 1024) < 1024) {
        maxLength = d_worklistLength - blockIdx.x * 1024 + 1;
        // To mark the boundary, maxLength is increased by 1 and the
        // numNeighbours of the last element is set to zero.
        temp[maxLength - 1] = 0;
    } else
        maxLength = 1024;
    if (blockIdx.x * blockDim.x + tId < d_worklistLength) {
        vertex = d_worklist[blockIdx.x * blockDim.x + tId];
        numNeighbours = graph[0][vertex + 1] - graph[0][vertex];
        temp[tId] = numNeighbours;
    }
    //print("This is a thread : %d. Max = %d\n", threadIdx.x, maxLength);
    __syncthreads();
    int index =  2 * tId, add = 1;
    for (int depth = maxLength; depth > 0; depth = depth >> 1) {
        if (index + add < maxLength) {
            temp[index] += temp[index + add];
            index = index << 1;
            add = add << 1;
        }
        __syncthreads();
    }
    if (tId == 0) {
        d_prefixLevel[blockIdx.x] = add;
        blockPrefixSum = temp[0];
        print("Level = %d. MaxLength = %d\n", d_prefixLevel[blockIdx.x], maxLength);
    }
    
    /*if (tId < d_worklistLength)
        d_prefixSum[tId] = temp[tId];*/

    __syncthreads();
    int level;
    level = d_prefixLevel[blockIdx.x];
    index = tId * level;
    for (int depth = maxLength; depth > 0; depth = depth >> 1) {
        if (index + level / 2 < maxLength) {
            temp[index] -= temp[index + level / 2];
            d_prefixSum[blockIdx.x * blockDim.x + index + level / 2] = temp[index] + d_prefixSum[blockIdx.x * blockDim.x + index];
        }
        index = index >> 1;
        level = level >> 1;
        __syncthreads();
    }
    if (tId == 0) {
        d_prefixLevel[blockIdx.x] = blockPrefixSum;
        print("Block %d. PrefixSum = %d, Array Value = %d\n", blockIdx.x, blockPrefixSum, d_prefixLevel[blockIdx.x]);
    }
}

// Prefix Sum on the whole block sum.
__global__ void Cuda_BlockOffsetPrefixSum(int numBlocks) {
    
    extern __shared__ int temp[];
    int tId = threadIdx.x;

    if (tId < numBlocks)
        temp[tId] = d_prefixLevel[tId];

    int index =  2 * tId, add = 1;
    __shared__ int sharedVar;
    for (int depth = numBlocks; depth > 0; depth = depth >> 1) {
        if (index + add < numBlocks) {
            temp[index] += temp[index + add];
            index = index << 1;
            add = add << 1;
        }
        __syncthreads();
    }
    if (tId == 0) {
        sharedVar = add;
        d_worklistLength = temp[0];
        print("WorkList Length = %d\n", d_worklistLength);
    }
    __syncthreads();
    
    int level;
    level = sharedVar;
    index = tId * level;
    for (int depth = numBlocks; depth > 0; depth = depth >> 1) {
        if (index + level / 2 < numBlocks) {
            temp[index] -= temp[index + level / 2];
            d_blockPrefixSum[blockIdx.x * blockDim.x + index + level / 2] = temp[index] + d_blockPrefixSum[blockIdx.x * blockDim.x + index];
        }
        index = index >> 1;
        level = level >> 1;
        __syncthreads();
    }
}

__global__ void Cuda_AddBlockPrefix() {

    int tId = blockIdx.x * blockDim.x + threadIdx.x;
    if (tId < d_worklistLength) {
        d_prefixSum[tId] += d_blockPrefixSum[blockIdx.x];
    }
}

int CudaGraphClass::verifyPrefixSum(int *calculatedPrefix) {
    
    int *verifiedPrefix, prefix = 0;
    verifiedPrefix = new int[(numVertices + 1)];
    for (int vertex = 0; vertex < numVertices; vertex++) {
        verifiedPrefix[vertex] = prefix;
        int numNeighbours = row[0][vertex + 1] - row[0][vertex];
        prefix += numNeighbours;
    }
    /*for (int vertex = 0; vertex <= numVertices; vertex++) {
        print("Prefix[%d] = %d\n", vertex, verifiedPrefix[vertex]);
    }*/
    for (int vertex = 0; vertex < numVertices; vertex++) {
        if (verifiedPrefix[vertex] != calculatedPrefix[vertex]) {
            print("Verification failed at vertex %d.\n", vertex);
            print("Verified prefix = %d. Calculated prefix = %d\n", verifiedPrefix[vertex], calculatedPrefix[vertex]);
            return 1;
        }
    }
    delete[] verifiedPrefix;
    return 0;
}

int CudaGraphClass::verifyGatherWorklist(int *calculatedGatherWorklist, int newWorklistLength) {

    int vertex = 0, i = 0;
    while (i < newWorklistLength) {
        for (int j = row[0][vertex]; j < row[0][vertex + 1]; j++, i++) {
            if (row[1][j] != calculatedGatherWorklist[i])
                return 1;
        }
        vertex++;
    }
    cout << "Verified: " << i << "\n";
    return 0;
}


inline int reallocDeviceMemory(int *d_pointer, int newMemorySize) {

    int *devicePointer;
    hipError_t err;

    err = hipMemcpyFromSymbol(&devicePointer, HIP_SYMBOL(d_pointer), sizeof(int *), 0, hipMemcpyDeviceToHost);
    CUDA_ERR_CHECK;
    hipFree(devicePointer);
    
    err = hipMalloc((void **)&devicePointer, newMemorySize * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_pointer), &devicePointer, sizeof(int *), 0, hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;

    return 0;
}

int CudaGraphClass::PrefixSum(int worklistLength, int *newWorklistLength) {

    hipError_t err;
    if (maxWorklistLength < worklistLength) {
        cout << "PrefixSum Realloc\n";
        int *devicePrefixSum, *newPrefixSum;
        maxWorklistLength = worklistLength;
        
        err = hipMemcpyFromSymbol(&devicePrefixSum, HIP_SYMBOL(d_prefixSum), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        hipFree(devicePrefixSum);
        
        err = hipMalloc((void **)&newPrefixSum, maxWorklistLength * sizeof(int));
        CUDA_ERR_CHECK;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_prefixSum), &newPrefixSum, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;

        //reallocDeviceMemory(d_worklist, maxWorklistLength);
        /*int *deviceWorklist, *newWorklist;
        err = hipMemcpyFromSymbol(&deviceWorklist, HIP_SYMBOL(d_worklist), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        hipFree(deviceWorklist);
        
        err = hipMalloc((void **)&newWorklist, maxWorklistLength * sizeof(int));
        CUDA_ERR_CHECK;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_worklist), &newWorklist, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;*/
    }
    int numBlocksPerGrid = (worklistLength + numThreadsPerBlock) / numThreadsPerBlock;
    if (maxNumBlocksPerGrid < numBlocksPerGrid) {
        maxNumBlocksPerGrid = numBlocksPerGrid;

        //reallocDeviceMemory(d_prefixLevel, maxNumBlocksPerGrid);
        int *devicePrefixLevel, *newPrefixLevel;
        err = hipMemcpyFromSymbol(&devicePrefixLevel, HIP_SYMBOL(d_prefixLevel), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        hipFree(devicePrefixLevel);
        
        err = hipMalloc((void **)&newPrefixLevel, maxNumBlocksPerGrid * sizeof(int));
        CUDA_ERR_CHECK;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_prefixLevel), &newPrefixLevel, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;

        //reallocDeviceMemory(d_blockPrefixSum, maxNumBlocksPerGrid);
        int *deviceBlockPrefixSum, *newBlockPrefixSum;
        err = hipMemcpyFromSymbol(&deviceBlockPrefixSum, HIP_SYMBOL(d_blockPrefixSum), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        hipFree(deviceBlockPrefixSum);
        
        err = hipMalloc((void **)&newBlockPrefixSum, maxNumBlocksPerGrid * sizeof(int));
        CUDA_ERR_CHECK;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_blockPrefixSum), &newBlockPrefixSum, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
    }

    Cuda_IntraBlockPrefixSum<<<numBlocksPerGrid, numThreadsPerBlock, numThreadsPerBlock * sizeof(int)>>>();
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    Cuda_BlockOffsetPrefixSum<<<(numBlocksPerGrid + numThreadsPerBlock) / numThreadsPerBlock, numThreadsPerBlock, numBlocksPerGrid * sizeof(int )>>>(numBlocksPerGrid);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    Cuda_AddBlockPrefix<<<numBlocksPerGrid, numThreadsPerBlock>>>();
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    int *devicePrefixSum, *hostPrefixSum;

    hostPrefixSum = new int[(worklistLength + 1)];
    out << "Prefix Sums\n";
    err = hipMemcpyFromSymbol(&devicePrefixSum, HIP_SYMBOL(d_prefixSum), sizeof(int *), 0, hipMemcpyDeviceToHost);
    CUDA_ERR_CHECK;
    err = hipMemcpy(hostPrefixSum, devicePrefixSum, (worklistLength + 1) * sizeof(int), hipMemcpyDeviceToHost);
    CUDA_ERR_CHECK;
    for (int i = 0; i <= worklistLength; i++)
        out << "["<< i << "] = " << hostPrefixSum[i] << endl;

    verifyPrefixSum(hostPrefixSum);
    err = hipMemcpyFromSymbol(&worklistLength, HIP_SYMBOL(d_worklistLength), sizeof(int), 0, hipMemcpyDeviceToHost);
    CUDA_ERR_CHECK;
    *newWorklistLength = worklistLength;
    delete[] hostPrefixSum;
    return 0;
}

// TODO: Build an optimized fine grained gathering algorithm
__global__ void populateNeighbours(int worklistLength) {

    int tId = blockIdx.x * blockDim.x + threadIdx.x;
    if (tId < worklistLength) {
        int vertex = d_worklist[tId];
        int edgeIndex = graph[0][vertex];
        int index = d_prefixSum[tId], lastIndex = d_prefixSum[tId + 1];
        //print("Thread: %d: vertex = %d, edgeIndex = %d, prefix = %d, lastIndex = %d\n", tId, vertex, edgeIndex, index, lastIndex);
        for (int i = 0; i < lastIndex - index; i++) {
            d_gatherWorklist[index + i] = graph[1][edgeIndex + i];
        }
    }
}

int CudaGraphClass::gatherNeighbours(int worklistLength) {

    int numBlocksPerGrid = (worklistLength + 1 + numThreadsPerBlock - 1) / numThreadsPerBlock;
    gpuErrchk(hipDeviceSynchronize());
    cout << "Gather Neighbours: " << numThreadsPerBlock << ", " << numBlocksPerGrid << ", " << worklistLength << "\n";
    populateNeighbours<<<numBlocksPerGrid, numThreadsPerBlock>>>(worklistLength);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    return 0;
}

int CudaGraphClass::callSSSP() {

    int terminate = false, *distance, *d_distance, worklistLength = numVertices;
    hipError_t err;

    distance = new int[(numVertices + 1)];
    err = hipMalloc((void **)&d_distance, (numVertices + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemset(d_distance, 0xff, (numVertices + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    //int numBlocksPerGrid = (numVertices + 1 + numThreadsPerBlock - 1) / numThreadsPerBlock;
    //cout << numThreadsPerBlock << ", " << numBlocksPerGrid << "\n";

    while (terminate == false) {
        terminate = true;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_terminate), &terminate, sizeof(bool), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        gpuErrchk(hipDeviceSynchronize());

        int newWorklistLength;
        PrefixSum(worklistLength, &newWorklistLength);

        if (maxWorklistLength < newWorklistLength) {
            maxWorklistLength = newWorklistLength;
            
            //reallocDeviceMemory(d_worklist, maxWorklistLength);
            cout << "After Prefix Sum: Realloc\n";
            int *deviceGatherWorklist, *newGatherWorklist;
            err = hipMemcpyFromSymbol(&deviceGatherWorklist, HIP_SYMBOL(d_gatherWorklist), sizeof(int *), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            hipFree(deviceGatherWorklist);
            
            err = hipMalloc((void **)&newGatherWorklist, maxWorklistLength * sizeof(int));
            CUDA_ERR_CHECK;
            err = hipMemcpyToSymbol(HIP_SYMBOL(d_gatherWorklist), &newGatherWorklist, sizeof(int *), 0, hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;

            delete[] distance;
            distance = new int[maxWorklistLength];
        }
        cout << "New WorkList in Host = " << newWorklistLength << "\n";
        gatherNeighbours(worklistLength);

        int *deviceWorklist, *deviceGatherWorklist;
        err = hipMemcpyFromSymbol(&deviceGatherWorklist, HIP_SYMBOL(d_gatherWorklist), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        err = hipMemcpyFromSymbol(&deviceWorklist, HIP_SYMBOL(d_worklist), sizeof(int *), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;

        err = hipMemcpy(distance, deviceGatherWorklist, newWorklistLength * sizeof(int), hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        cout << "New Worklist: \n";
        for (int i = 0; i < newWorklistLength; i++)
            out << "["<< i << "] = " << distance[i] << endl;

        if (verifyGatherWorklist(distance, newWorklistLength) == 1)
            cout << "Verification Failed\n";

        err = hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(d_terminate), sizeof(bool), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;

        worklistLength = newWorklistLength;
        // Swap worklist and gatherWorklist
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_worklist), &deviceGatherWorklist, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_gatherWorklist), &deviceWorklist, sizeof(int *), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
    }
    
    delete[] distance;
    return 0;
}


int CudaGraphClass::copyGraphToDevice() {

    CUDA_SET_DEVICE_ID;
    gpuErrchk(hipPeekAtLastError());
    int *vertexArray, *edgeArray, *weightArray, *worklist, *worklist2;
    int *prefixSum, *blockPrefixSum, *prefixLevel;
    hipError_t err;
    err = hipMalloc((void **)&vertexArray, (numVertices + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&edgeArray, (numEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&weightArray, (numEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    maxWorklistLength = numVertices + 2;
    err = hipMalloc((void **)&worklist, maxWorklistLength * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&worklist2, maxWorklistLength * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&prefixSum, maxWorklistLength * sizeof(int));
    CUDA_ERR_CHECK;
    maxNumBlocksPerGrid = 1024;
    err = hipMalloc((void **)&blockPrefixSum, maxNumBlocksPerGrid * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&prefixLevel, maxNumBlocksPerGrid * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemset(prefixSum, 0x0, maxWorklistLength * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemset(blockPrefixSum, 0x0, maxNumBlocksPerGrid * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemcpy(vertexArray, row[0], (numVertices + 2) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    err = hipMemcpy(edgeArray, row[1], (numEdges + 1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    err = hipMemcpy(weightArray, row[2], (numEdges + 1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    CudaInitialize<<<1, 1>>>(vertexArray, edgeArray, weightArray, worklist, worklist2, prefixSum, blockPrefixSum, prefixLevel, numVertices, numEdges);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    return 0;
}

void CudaGraphClass::populate(char *fileName) {

    inputFile.open(fileName);
    if (!inputFile.is_open()){
        cout << "invalid file";
        return;
    }

    int **AdjMatrix, i, j, k;
    AdjMatrix = new int* [numVertices + 1]();
    for (i = 0; i <= numVertices; i++) {

        AdjMatrix[i] = new int [numVertices + 1]();
    }
    i = numEdges;
    int lastj = 0, currentIndex = 0;
    inputFile >> j >> k;
    srand(time(NULL));
    while(i > 0) {

        //scanf("%d %d", &j, &k);
        inputFile >> j >> k;
        AdjMatrix[j][k] = 1;
        while (lastj <= j || lastj == 0) {
            if (lastj == 0) {
                row[0][0] = currentIndex;
                row[0][1] = currentIndex;
            }else {
                row[0][lastj] = currentIndex;
            }
            lastj++;
        }
//        if (AdjMatrix[k][j] != 1)
        row[1][currentIndex] = k;
        row[2][currentIndex] = (rand() % 2) ? rand() % 10 - 10 : rand() % 10;
        currentIndex ++;
        i--;
    }
    //row[1][0] = 0;
    // Sentinel node just points to the end of the last node in the graph
    while (lastj <= numVertices + 1) {
        row[0][lastj] = currentIndex;
        lastj++;
    }
    //row[0][lastj+1] = currentIndex;
/*    for (i = 0; i <= numVertices + 1; i++)
        print("Vertex: %d = %d\n", i, row[0][i]);

    print("Second Array:\n");
    for (i = 0; i <= numEdges; i++)
        print("Edges: Index: %d, Value = %d\n", i, row[1][i]);
*/
    j = 1;
    for (i = 1; i <= numVertices; i++) {

        currentIndex = row[0][i];
        while (currentIndex < row[0][i+1]) {
//            print("%d %d\n", i, row[1][currentIndex]);
            if (AdjMatrix[i][row[1][currentIndex]] != 1 /*&&
                AdjMatrix[row[1][currentIndex]][i] != 1*/) {
                outs("\n\nGraph Do not Match at [%d][%d]. CurrentIndex = %d\n\n", i, row[1][currentIndex], currentIndex);
                break;
            }
            j++;
            currentIndex ++;
        }
    }
    for (i = 0; i <= numVertices; i++) {

        delete[] AdjMatrix[i];
    }
    delete[] AdjMatrix;
}
void CudaGraphClass::printGraph() {
    CudaPrintGraph<<<1, 1>>>();
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}
