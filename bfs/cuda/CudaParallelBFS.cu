#include "hip/hip_runtime.h"
#include <iostream>
#include "CudaGraph.h"
#define cprint(...) printMutex.lock(); printf(__VA_ARGS__); printMutex.unlock();
#define p(x) printMutex.lock(); cout << x << endl; printMutex.unlock();

#define CUDA_ERR_CHECK  \
if( err != hipSuccess) { \
    printf("CUDA error: %s ** at Line %d\n", hipGetErrorString(err), __LINE__); \
    return EXIT_FAILURE; \
} \

using namespace std;

__device__ int graph[2], *worklist, d_tail;
__device__ bool d_terminate;
__global__ void Cuda_BFS(CudaGraphClass *graphData) {
    
    printf("BlockId = %d, Thread ID : %d\n", blockIdx.x, threadIdx.x);
    int tId = blockIdx.x * blockDim.x + threadIdx.x;
}

void CudaGraphClass::callBFS() {

    print("Hello inside cuda code\n");
    copyGraphToDevice();
    
    int terminate = false;
    while (terminate == false) {
        terminate = true;
        outs("Queue: Head: %d, Tail: %d\n", currentQueueHead, currentQueueTail);
        hipMemcpyToSymbol(HIP_SYMBOL(d_terminate), &terminate, sizeof(bool), 0, hipMemcpyHostToDevice);
//        hipMemcpyToSymbol(HIP_SYMBOL(d_tail), &tail, sizeof(bool), 0, hipMemcpyHostToDevice);
        Cuda_BFS<<<2, 5>>>(this);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&terminate, HIP_SYMBOL(d_terminate), sizeof(bool), 0, hipMemcpyDeviceToHost);
    }
}
int CudaGraphClass::copyGraphToDevice() {

    hipError_t err;
    err = hipMalloc((void **)&graph[0], (numVertices + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&graph[1], (numEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMemcpy(graph[0], row[0], (numVertices + 2) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    err = hipMemcpy(graph[1], row[1], (numEdges + 1) * sizeof(int), hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    return 0;
}
